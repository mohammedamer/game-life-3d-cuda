
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

struct index {
  int x;
  int y;
  int z;
};

__device__
struct index unravel_idx(int idx, int n){

  struct index unravel;

  int x, y, z;

  x = idx / (n*n);
  y = (idx / n) % n;
  z = idx % n;

  unravel = {.x = x, .y = y, .z = z};

  return unravel;

} 

__device__
int ravel_idx(struct index idx, int n){
  return idx.x+n*(idx.y + (idx.z*n));
}

__device__
int should_live(int is_alive, int alive_count){

  if (is_alive == 1){

    if (alive_count < 4 || alive_count > 5){
      return 0;
    }

  } else{

    if (alive_count == 5){
      return 1;
    }

  }

  return is_alive;

}

__global__
void evolve_kernel(int *cell_arr, int *out_arr, int n)
{
  int num_elem = n*n*n;

  int current_idx = blockIdx.x*blockDim.x+threadIdx.x;

  for (int idx=current_idx;
    idx<num_elem; 
    idx+=blockDim.x*gridDim.x){

      struct index idx_3d = unravel_idx(idx, n);

      int alive_count = 0;

      int adj_x[] = {idx_3d.x-1, idx_3d.x, idx_3d.x+1};

      int adj_y[] = {idx_3d.y-1, idx_3d.y, idx_3d.y+1};

      int adj_z[] = {idx_3d.z-1, idx_3d.z, idx_3d.z+1};

      for (int i = 0; i < 3; i++){
        for (int j = 0; j < 3; j++){
          for (int k=0; k<3; k++){

            struct index _idx;

            _idx.x = adj_x[i];
            _idx.y = adj_y[j];
            _idx.z = adj_z[k];

            int adj_idx = ravel_idx(_idx, n);

            if (adj_idx != idx && adj_idx > 0 && adj_idx < num_elem ){
              alive_count+=cell_arr[adj_idx];
            } 

          }
        }
      }

      int is_alive = cell_arr[current_idx];

      out_arr[current_idx] = should_live(is_alive, alive_count);

    }

}

void evolve(int *cell_arr, int *out_arr, int n)
{
  
  int *_in, *_out;
  int num_elem;

  num_elem = n*n*n;

  hipMallocManaged(&_in, num_elem*sizeof(int));
  hipMallocManaged(&_out, num_elem*sizeof(int));

  for (int i = 0; i < num_elem; i++) {
    _in[i] = cell_arr[i];
  }

  evolve_kernel<<<16,256>>>(_in, _out, n);

  hipDeviceSynchronize();

  hipFree(_in);

  for (int i = 0; i < num_elem; i++) {
    out_arr[i] = _out[i];
  }

  hipFree(_out);
}