
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

struct index {
  int x;
  int y;
  int z;
};

__device__
struct index unravel_idx(int idx, int n){

  struct index unravel;

  int x, y, z;

  z = idx % n;
  y = (idx / n) % n;
  x = (idx / n) / n;

  unravel = {.x = x, .y = y, .z = z};

  return unravel;

} 

__device__
int ravel_idx(struct index idx, int n){
  return idx.x*n*n+idx.y*n + idx.z;
}

__device__
int should_live(int is_alive, int alive_count){

  if (is_alive == 1){

    if (alive_count < 4 || alive_count > 5){
      return 0;
    }

  } else{

    if (alive_count == 5){
      return 1;
    }

  }

  return is_alive;

}

__global__
void evolve_kernel(int *cell_arr, int *out_arr, int n)
{
  int num_elem = n*n*n;

  // cell index
  int current_idx = blockIdx.x*blockDim.x+threadIdx.x;

  // grid-stride loop

  for (int idx=current_idx;
    idx<num_elem; 
    idx+=blockDim.x*gridDim.x){

      struct index idx_3d = unravel_idx(idx, n);

      int alive_count = 0;

      // collect adjacent

      int adj_x[] = {idx_3d.x-1, idx_3d.x, idx_3d.x+1};

      int adj_y[] = {idx_3d.y-1, idx_3d.y, idx_3d.y+1};

      int adj_z[] = {idx_3d.z-1, idx_3d.z, idx_3d.z+1};

      // count live

      for (int i = 0; i < 3; i++){
        for (int j = 0; j < 3; j++){
          for (int k=0; k<3; k++){

            struct index _idx;

            _idx.x = adj_x[i];
            _idx.y = adj_y[j];
            _idx.z = adj_z[k];

            if ((_idx.x > -1 && _idx.x < n) &&
            (_idx.y > -1 && _idx.y < n) &&
            (_idx.z > -1 && _idx.z < n)){
              
              int adj_idx = ravel_idx(_idx, n);

              if (adj_idx != idx){
                alive_count+=cell_arr[adj_idx];
              } 
            }

          }
        }
      }

      int is_alive = cell_arr[current_idx];

      // apply local rules

      out_arr[current_idx] = should_live(is_alive, alive_count);

    }

}

void evolve(int *cell_arr, int *out_arr, int n)
{
  
  int *_in, *_out;
  int num_elem;

  num_elem = n*n*n;

  // allocate unified mem

  hipMallocManaged(&_in, num_elem*sizeof(int));
  hipMallocManaged(&_out, num_elem*sizeof(int));

  // copy input

  for (int i = 0; i < num_elem; i++) {
    _in[i] = cell_arr[i];
  }

  // call kernel

  int threadsPerBlock = 256;
  int blocks = (num_elem+threadsPerBlock)/threadsPerBlock;

  evolve_kernel<<<blocks,threadsPerBlock>>>(_in, _out, n);

  // sync with GPU

  hipDeviceSynchronize();

  hipFree(_in);

  // copy output

  for (int i = 0; i < num_elem; i++) {
    out_arr[i] = _out[i];
  }

  hipFree(_out);
}